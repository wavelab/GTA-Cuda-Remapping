#include "hip/hip_runtime.h"
#include <exception>
#include <ctime>
#include <stdlib.h>
#include <sstream>
#include <string>
#include <random>
#include <chrono>
#include <regex>
#include <experimental/filesystem>
#include <hip/hip_runtime_api.h>
#include <opencv2/opencv.hpp>
#include <thread>
#include <mutex>
#include <queue>
#include <boost/lockfree/queue.hpp>

#include "GpuMat.cuh"
#include "GpuVector.cuh"
#include "helpers.h"
#include "colours.h"
namespace fs = std::experimental::filesystem;

//std::atomic_bool done = false;
//boost::lockfree::queue<cv::Mat> matQueue(2000);

// void readThread(std::vector<std::string>& all_files)
// {
// 	for(int i = 0; i < all_files.size(); i++)
// 	{
// 		std::cout << "reading " << all_files[i] << std::endl;
// 		cv::Mat img = cv::imread(all_files[i], CV_LOAD_IMAGE_COLOR);
// 		matQueue.push(img);
// 	}
// 	done = true;
// }

const int device = 2;

bool hasEnding (std::string const &fullString, std::string const &ending) {
    if (fullString.length() >= ending.length()) {
        return (0 == fullString.compare (fullString.length() - ending.length(), ending.length(), ending));
    } else {
        return false;
    }
}

void processFiles(std::vector<std::string> files, std::string image_path, std::string output_path, std::string outfile)
{
	hipSetDevice(device);
	auto& fout = std::cout;
	//std::ofstream fout(outfile, std::ofstream::out | std::ofstream::app);
	std::unique_ptr<GpuMat<unsigned char>> scratchGpuMat;
	std::unique_ptr<GpuMat<unsigned char>> outputImg;

	float readTime = 0.f;

	auto startTime = std::chrono::high_resolution_clock::now(); //to beat 58 s
	for(int i = 0; i < files.size(); i++)
	{
		auto readStart = std::chrono::high_resolution_clock::now();
		fout << "reading " << files[i] << std::endl;
		cv::Mat img;
		while(!img.data)
		{
			try
			{
				img = cv::imread(files[i], CV_LOAD_IMAGE_COLOR);
			}
			catch(std::exception& ex)
			{
				fout << "caught a thing" << std::endl;
			}
			if(!img.data) //only be able to parse if IEND chunk is found (i.e. transer complete)
				std::this_thread::sleep_for(std::chrono::milliseconds(100));
		}
		auto readEnd = std::chrono::high_resolution_clock::now();

		if(i == 0)
		{
			scratchGpuMat = std::unique_ptr<GpuMat<unsigned char>>(new GpuMat<unsigned char>(img.rows, img.cols, img.channels(), false));//do this to allocate memory
			outputImg = std::unique_ptr<GpuMat<unsigned char>>(new GpuMat<unsigned char>(img.rows, img.cols, img.channels(), false));
		}
		scratchGpuMat->load(img);
		scratchGpuMat->mapColours(*outputImg); //(GpuMat<dtype>& to, GpuVector<dtype>& map)
		
		cv::Mat outMat = outputImg->getMat();

		std::string outFileName = std::regex_replace(files[i], std::regex(image_path), output_path);
		fs::create_directories(fs::path(outFileName).parent_path());
		fout << "writing " << outFileName << std::endl;
		cv::imwrite(outFileName, outMat);

		readTime += (float)std::chrono::duration_cast<std::chrono::milliseconds>(readEnd - readStart).count()/1000.f;
	}
	auto endTime = std::chrono::high_resolution_clock::now();

	float totalTime = (float)std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count()/1000.0f;

	fout << files.size() << std::endl;
	fout << "total time: " << totalTime << " seconds" << std::endl;
	fout << "read time: " << readTime << " seconds" << std::endl;
}

template<typename T>
std::vector<std::vector<T>> SplitVector(const std::vector<T>& vec, size_t n)
{
    std::vector<std::vector<T>> outVec;

    size_t length = vec.size() / n;
    size_t remain = vec.size() % n;

    size_t begin = 0;
    size_t end = 0;

    for (size_t i = 0; i < std::min(n, vec.size()); ++i)
    {
        end += (remain > 0) ? (length + !!(remain--)) : length;

        outVec.push_back(std::vector<T>(vec.begin() + begin, vec.begin() + end));

        begin = end;
    }

    return outVec;
}

std::vector<std::string> GetImagesToProcess(std::string& inputPath, std::string& outputPath)
{
	std::vector<std::string> ret;
	for(auto& p: fs::recursive_directory_iterator(inputPath))
	{
		std::string curPath = p.path().string();
		bool regFile = fs::is_regular_file(p);
		if(regFile && hasEnding(curPath, "png"))
		{
			std::string imgPath = std::regex_replace(curPath, std::regex(inputPath), outputPath);

			if(!fs::is_regular_file(imgPath))
			{
				ret.push_back(curPath);
			}
		}
	}
	return ret;
}

/**
 * contains cuda specific initializations
 */
int main(int argc, char** argv )
{	
	// grab the arguments
	std::string image_path;
	std::string output_path;
	int numProc = 8;
	std::vector<int> availGpu = {0,1,2};
	for (int i = 0; i < argc; i++)
	{
		if (strcmp(argv[i], "-i") == 0)
			image_path = argv[i+1];
		if (strcmp(argv[i], "-o") == 0)
			output_path = argv[i+1];
		if (strcmp(argv[i], "-n") == 0)
			numProc = atoi(argv[i+1]);
	}

	// std::vector<std::string> all_files;

	// for(auto& p: fs::recursive_directory_iterator(image_path))
	// {
	// 	std::string val = p.path().string();
	// 	std::string fname = p.path().filename().string();
	// 	if(Helpers::hasEnding(val,"png") && fname[0] == 'n')
	// 		all_files.push_back(val);
	// }

	// std::cout << "found " << all_files.size() << " images" << std::endl;

	// std::vector<std::vector<std::string>> chunks = SplitVector(all_files, numProc);
	hipSetDevice(device);

	for(;;)
	{
		std::vector<std::string> toProcess = GetImagesToProcess(image_path, output_path);
		if(toProcess.size() > 0)
		{
			std::vector<std::vector<std::string>> chunks = SplitVector(toProcess, numProc);

			std::vector<std::thread> threads;

			for(int i = 0; i < numProc && i < chunks.size(); i++)
			{
				std::stringstream ss;
				ss << i << ".out";
				threads.push_back(std::thread(processFiles, chunks[i], image_path, output_path, ss.str()));
			}

			for(std::thread& t : threads)
			{
				t.join();
			}
		}
		std::this_thread::sleep_for(std::chrono::milliseconds(5000));
	}

	//return 0;
}
